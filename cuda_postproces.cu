#include "hip/hip_runtime.h"
// Taken originally from
// https://www.3dgep.com/opengl-interoperability-with-cuda/
// by Jeremiah van Oosten, 2011
// https://drive.google.com/file/d/0B0ND0J8HHfaXT0p1N3ZkSW5kTVU/edit?usp=sharing

// Updates over the original version:
// - Updated to CUDA 11, 64b, removed unsupported dependencies
// - Moved to use OpenGL 4
// - 2-pass rendering shader controls
// - Separated the GL rendering and CUDA code-base
// - Added debug reads for debugging offscreen rendering
// - Compilable with Visual Studio 2019 142 toolsets
//   Added device to GL copy code (CUDACopyDeviceToGL function)

#include <hip/hip_runtime_api.h>
#include "cutil.h"
#include "cutil_inline_runtime.h"
#include "cuda-postprocess.h"

#define USE_SHARED_MEM 0

#define FILTER_SIZE (5*5) // 5x5 kernel filter
#define BLOCK_SIZE 16     // block size

__device__ __constant__ float kernelFilter_D[FILTER_SIZE];
__device__ __constant__ int indexOffsetsU_D[25];
__device__ __constant__ int indexOffsetsV_D[25];
__device__ __constant__ float invScale_D;
__device__ __constant__ float offset_D;

texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef;

template< typename R, typename T >
__device__ R Clamp( T value, T min, T max )
{
    if ( value < min )
    {
        return (R)min;
    }
    else if ( value > max )
    {
        return (R)max;
    }
    else
    {
        return (R)value;
    }
}

__global__ void PostprocessKernel( uchar4* dst, unsigned int imgWidth, unsigned int imgHeight )
{
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int bw = blockDim.x;
    unsigned int bh = blockDim.y;
    // Non-normalized U, V coordinates of input texture for current thread.
    unsigned int u = ( bw * blockIdx.x ) + tx;
    unsigned int v = ( bh * blockIdx.y ) + ty;

    // Early-out if we are beyond the texture coordinates for our texture.
    if ( u > imgWidth || v > imgHeight ) return;

#if USE_SHARED_MEM
    __shared__ uchar4 sTex[BLOCK_SIZE+4][BLOCK_SIZE+4]; // 20 * 20 * 4 Bytes = 1,600 Bytes ~= 1.5 KB
    // U, V, coordinates relative to the shared memory block
    unsigned int sU = tx + 2;
    unsigned int sV = ty + 2;

    // Load the current (center) pixel into shared memory
    sTex[sU][sV] = tex2D( texRef, u, v );
    if ( tx < 2 )
    {
        // Left-edge
        sTex[tx][sV] = tex2D( texRef, u - 2, v );
        // Right-edge
        sTex[BLOCK_SIZE+sU][sV] = tex2D( texRef, u + BLOCK_SIZE, v );
    }
    if ( ty < 2 )
    {
        // Top-edge
        sTex[sU][ty] = tex2D( texRef, u, v - 2 );
        // Bottom-edge
        sTex[sU][BLOCK_SIZE+sV] = tex2D( texRef, u, v + BLOCK_SIZE );
    }
    if ( tx < 2 && ty < 2 ) // Corners
    {
        // Top-left 
        sTex[tx][ty] = tex2D(texRef, u - 2, v - 2 );
        // Top-right
        sTex[BLOCK_SIZE + sU][ty] = tex2D( texRef, u + BLOCK_SIZE, v - 2 );

        // Bottom-left
        sTex[tx][BLOCK_SIZE + sV] = tex2D( texRef, u - 2, v + BLOCK_SIZE );
        // Bottom-right
        sTex[BLOCK_SIZE + sU][BLOCK_SIZE + sV] = tex2D( texRef, u + BLOCK_SIZE, v + BLOCK_SIZE );
    }
    __syncthreads();
#endif

    unsigned int index = ( v * imgWidth ) + u;
    
    float4 tempColor = make_float4(0, 0, 0, 1);
    for ( int i = 0; i < FILTER_SIZE; ++i )
    {
#if USE_SHARED_MEM
        uchar4 color = sTex[sU + indexOffsetsU_D[i]][sV + indexOffsetsV_D[i]]; 
#else
        uchar4 color = tex2D( texRef, u + indexOffsetsU_D[i], v + indexOffsetsV_D[i] );
#endif
        tempColor.x += color.x * kernelFilter_D[i];
        tempColor.y += color.y * kernelFilter_D[i];
        tempColor.z += color.z * kernelFilter_D[i];
    }

    dst[index] = make_uchar4( Clamp<unsigned char>(tempColor.x * invScale_D + offset_D, 0.0f, 255.0f), Clamp<unsigned char>(tempColor.y * invScale_D + offset_D, 0.0f, 255.0f), Clamp<unsigned char>(tempColor.z * invScale_D + offset_D, 0.0f, 255.0f), 1 );
}

// Copy CUDA BGRA buffer directly to OpenGL
void CUDACopyDeviceToGL( hipGraphicsResource_t& dstGLDeviceBuffer, 
                void* srcCudaDeviceBuffer,
                unsigned int sizeBytes )
{
    hipGraphicsResource_t resources[1] = { dstGLDeviceBuffer };

    // Map the resources so they can be used in the kernel.
    cutilSafeCall( hipGraphicsMapResources( 1, resources ) );

    hipArray* dstArray;

    // Get a device pointer to the OpenGL buffer
    cutilSafeCall( hipGraphicsSubResourceGetMappedArray( &dstArray, dstGLDeviceBuffer, 0, 0 ) );

    // Copy the destination back to the source array
    cutilSafeCall( hipMemcpyToArray( dstArray, 0, 0, srcCudaDeviceBuffer, sizeBytes, hipMemcpyDeviceToDevice ) );

    // Unmap the resources again so the texture can be rendered in OpenGL
    cutilSafeCall( hipGraphicsUnmapResources( 1, resources ) );
}

uchar4* g_dstBuffer = NULL;
size_t g_BufferSize = 0; 

void PostprocessCUDA( hipGraphicsResource_t& dst, hipGraphicsResource_t& src, unsigned int width, unsigned int height, float* filter_H, float scale, float offset_H )
{
    // Avoid divide by zero error:
    float invScale_H = ( scale == 0.0f ) ? 1.0f : 1.0f / scale;

    int indexOffsetsU_H[] = {
        -2, -1, 0, 1, 2,
        -2, -1, 0, 1, 2,
        -2, -1, 0, 1, 2,
        -2, -1, 0, 1, 2,
        -2, -1, 0, 1, 2,
    };
    int indexOffsetsV_H[] = {
        -2, -2, -2, -2, -2,
        -1, -1, -1, -1, -1,
         0,  0,  0,  0,  0,
         1,  1,  1,  1,  1,
         2,  2,  2,  2,  2,
    };
        
    // Copy the scale and offset to the device for use by the kernel.
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( invScale_D), &invScale_H, sizeof(float), 0, hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( offset_D), &offset_H, sizeof(float), 0, hipMemcpyHostToDevice) );
    
    // Copy the data in the filter to the constant device variable.
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( kernelFilter_D), filter_H, FILTER_SIZE * sizeof(float), 0, hipMemcpyHostToDevice ) );

    // Copy the index offset arrays to constant memory 
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( indexOffsetsU_D), indexOffsetsU_H, 25 * sizeof(int), 0, hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( indexOffsetsV_D), indexOffsetsV_H, 25 * sizeof(int), 0, hipMemcpyHostToDevice) );

    hipGraphicsResource_t resources[2] = { src, dst };

    // Map the resources so they can be used in the kernel.
    cutilSafeCall( hipGraphicsMapResources( 2, resources ) );

    hipArray* srcArray;
    hipArray* dstArray;

    // Get a device pointer to the OpenGL buffers
    cutilSafeCall( hipGraphicsSubResourceGetMappedArray( &srcArray, src, 0, 0 ) );
    cutilSafeCall( hipGraphicsSubResourceGetMappedArray( &dstArray, dst, 0, 0 ) );

    // Map the source texture to a texture reference.
    cutilSafeCall( hipBindTextureToArray( texRef, srcArray ) );

    // Destination buffer to store the result of the postprocess effect.
    size_t bufferSize = width * height * sizeof(uchar4);
    if ( g_BufferSize != bufferSize )
    {
        if ( g_dstBuffer != NULL )
        {
            hipFree( g_dstBuffer );
        }
        // Only re-allocate the global memory buffer if the screen size changes, 
        // or it has never been allocated before (g_BufferSize is still 0)
        g_BufferSize = bufferSize;
        cutilSafeCall( hipMalloc( &g_dstBuffer, g_BufferSize ) );
    }

    // Compute the grid size
    size_t blocksW = (size_t)ceilf( width / (float)BLOCK_SIZE );
    size_t blocksH = (size_t)ceilf( height / (float)BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

    PostprocessKernel<<< gridDim, blockDim >>>( g_dstBuffer, width, height );

    // Copy the destination back to the source array
    cutilSafeCall( hipMemcpyToArray( dstArray, 0, 0, g_dstBuffer, bufferSize, hipMemcpyDeviceToDevice ) );

    // Unbind the texture reference
    cutilSafeCall( hipUnbindTexture( texRef ) );

    // Unmap the resources again so the texture can be rendered in OpenGL
    cutilSafeCall( hipGraphicsUnmapResources( 2, resources ) );
}